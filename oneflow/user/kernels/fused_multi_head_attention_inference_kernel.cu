
#include <hip/hip_runtime.h>
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#ifdef WITH_CUTLASS

#include "oneflow/core/framework/framework.h"
#include "oneflow/core/ep/cuda/cuda_stream.h"
#include "oneflow/core/ep/include/primitive/permute.h"
#include "cutlass/arch/mma.h"
#include "cutlass/gemm/warp/mma.h"
#include "kernel_forward.h"
#include "oneflow/core/kernel/cuda_graph_support.h"
#include "trt_flash_attention/fmha.h"
#include "trt_flash_attention/fmha_flash_attention.h"

namespace oneflow {

namespace user_op {

namespace {

template<typename T, int pack_size>
struct alignas(pack_size * sizeof(T)) Pack {
  T elem[pack_size];
};

template<typename T>
__global__ void PackQkv(int b, int s, int nh, int d, const T* q, const T* k, const T* v, T* o,
                        int32_t* seq_len) {
  int count = b * s * nh * d * 3;
  for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < count; i += blockDim.x * gridDim.x) {
    int row = i / (d * 3);
    int out_col = i - row * (d * 3);
    T out;
    if (out_col < d) {
      out = q[row * d + out_col];
    } else if (out_col < 2 * d) {
      out = k[row * d + out_col - d];
    } else {
      out = v[row * d + out_col - d * 2];
    }
    o[i] = out;
  }
  for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < b + 1; i += blockDim.x * gridDim.x) {
    seq_len[i] = i * s;
  }
}

struct Params {
  DataType data_type;
  int64_t num_batches;
  int64_t num_heads;
  int64_t query_seq_len;
  int64_t kv_seq_len;
  int64_t head_size;
  int64_t value_head_size;
  int64_t q_stride_b;
  int64_t q_stride_m;
  int64_t q_stride_h;
  int64_t k_stride_b;
  int64_t k_stride_m;
  int64_t k_stride_h;
  int64_t v_stride_b;
  int64_t v_stride_m;
  int64_t v_stride_h;
  bool causal;
  int64_t causal_diagonal_offset;
  const void* query_ptr;
  const void* key_ptr;
  const void* value_ptr;
  const void* attn_bias_ptr;
  int64_t attn_bias_stride_b;
  int64_t attn_bias_stride_h;
  int64_t attn_bias_stride_m;
  void* out_ptr;
  void* workspace;
  int64_t workspace_size;
};

template<typename T, typename ArchTag, bool is_aligned, int queries_per_block, int keys_per_block,
         bool single_value_iteration, bool with_attn_bias>
void LaunchCutlassFmha(const Params& params, ep::CudaStream* stream) {
  // The fmha implementation below is based on xformers's fmha
  // implementation at:
  // https://github.com/facebookresearch/xformers/tree/main/xformers/csrc/attention/cuda/fmha
  using Attention = AttentionKernel<T, ArchTag, is_aligned, queries_per_block, keys_per_block,
                                    single_value_iteration, false, with_attn_bias>;
  typename Attention::Params p{};
  p.query_ptr = const_cast<T*>(reinterpret_cast<const T*>(params.query_ptr));
  p.key_ptr = const_cast<T*>(reinterpret_cast<const T*>(params.key_ptr));
  p.value_ptr = const_cast<T*>(reinterpret_cast<const T*>(params.value_ptr));
  p.attn_bias_ptr = const_cast<T*>(reinterpret_cast<const T*>(params.attn_bias_ptr));
  p.logsumexp_ptr = nullptr;
  p.output_ptr = reinterpret_cast<T*>(params.out_ptr);
  if (Attention::kNeedsOutputAccumulatorBuffer) {
    using Acc = typename Attention::accum_t;
    CHECK_GE(params.workspace_size, params.num_batches * params.query_seq_len * params.num_heads
                                        * params.value_head_size * sizeof(Acc));
    p.output_accum_ptr = reinterpret_cast<Acc*>(params.workspace);
  } else {
    p.output_accum_ptr = nullptr;
  }
  p.num_heads = params.num_heads;
  p.num_batches = params.num_batches;
  p.head_dim = params.head_size;
  p.head_dim_value = params.value_head_size;
  p.num_queries = params.query_seq_len;
  p.num_keys = params.kv_seq_len;
  p.q_strideM = params.q_stride_m;
  p.k_strideM = params.k_stride_m;
  p.v_strideM = params.v_stride_m;
  p.o_strideM = p.head_dim_value * p.num_heads;
  p.bias_strideM = params.attn_bias_stride_m;

  p.q_strideH = params.q_stride_h;
  p.k_strideH = params.k_stride_h;
  p.v_strideH = params.v_stride_h;
  p.bias_strideH = params.attn_bias_stride_h;

  p.q_strideB = params.q_stride_b;
  p.k_strideB = params.k_stride_b;
  p.v_strideB = params.v_stride_b;
  p.bias_strideB = params.attn_bias_stride_b;

  p.scale = 1.0 / std::sqrt(float(p.head_dim));

  p.causal = params.causal;
  p.causal_diagonal_offset = params.causal_diagonal_offset;
  p.use_dropout = false;

  constexpr auto kernel_fn = attention_kernel_batched_impl<Attention>;
  int smem_bytes = sizeof(typename Attention::SharedStorage);
  if (smem_bytes > 0xc000) {
    static bool once = [&]() {
      cudaFuncSetAttribute(kernel_fn, cudaFuncAttributeMaxDynamicSharedMemorySize, smem_bytes);
      return true;
    }();
  }
  CHECK(Attention::check_supported(p));
  kernel_fn<<<p.getBlocksGrid(), p.getThreadsGrid(), smem_bytes, stream->cuda_stream()>>>(p);
}

template<typename T, typename ArchTag, bool is_aligned, int queries_per_block, int keys_per_block,
         bool single_value_iteration>
void DispatchWithAttnBias(const Params& params, ep::CudaStream* stream) {
  if (params.attn_bias_ptr != nullptr) {
    LaunchCutlassFmha<T, ArchTag, is_aligned, queries_per_block, keys_per_block,
                      single_value_iteration, true>(params, stream);
  } else {
    LaunchCutlassFmha<T, ArchTag, is_aligned, queries_per_block, keys_per_block,
                      single_value_iteration, false>(params, stream);
  }
}

template<typename T, typename ArchTag, bool is_aligned, int queries_per_block, int keys_per_block>
void DispatchSingleValueIteration(const Params& params, ep::CudaStream* stream) {
  if (params.value_head_size <= keys_per_block) {
    DispatchWithAttnBias<T, ArchTag, is_aligned, queries_per_block, keys_per_block, true>(params,
                                                                                          stream);
  } else {
    DispatchWithAttnBias<T, ArchTag, is_aligned, queries_per_block, keys_per_block, false>(params,
                                                                                           stream);
  }
}

template<typename T, typename ArchTag, bool is_aligned>
void DispatchKeysPerBlock(const Params& params, ep::CudaStream* stream) {
  if (params.value_head_size <= 64) {
    DispatchSingleValueIteration<T, ArchTag, is_aligned, 64, 64>(params, stream);
  } else {
    DispatchSingleValueIteration<T, ArchTag, is_aligned, 32, 128>(params, stream);
  }
}

template<typename T, typename ArchTag>
void DispatchIsAligned(const Params& params, ep::CudaStream* stream) {
  if (reinterpret_cast<uintptr_t>(params.query_ptr) % 16 == 0
      && reinterpret_cast<uintptr_t>(params.key_ptr) % 16 == 0
      && reinterpret_cast<uintptr_t>(params.value_ptr) % 16 == 0
      && params.attn_bias_stride_m % (16 / sizeof(T)) == 0
      && params.head_size % (16 / sizeof(T)) == 0
      && params.value_head_size % (16 / sizeof(T)) == 0) {
    DispatchKeysPerBlock<T, ArchTag, true>(params, stream);
  } else {
    DispatchKeysPerBlock<T, ArchTag, false>(params, stream);
  }
}

template<typename T>
void DispatchArchTag(const Params& params, ep::CudaStream* stream) {
  const int major = stream->device_properties().major;
  const int minor = stream->device_properties().minor;

  if (major == 8) {
    DispatchIsAligned<T, cutlass::arch::Sm80>(params, stream);
  } else if (major == 7) {
    if (minor == 5) {
      DispatchIsAligned<T, cutlass::arch::Sm75>(params, stream);
    } else {
      DispatchIsAligned<T, cutlass::arch::Sm70>(params, stream);
    }
  } else {
    UNIMPLEMENTED();
  }
}

void DispatchCutlassFmha(const Params& params, ep::CudaStream* stream) {
  if (params.data_type == DataType::kFloat16) {
    DispatchArchTag<cutlass::half_t>(params, stream);
  } else if (params.data_type == DataType::kFloat) {
    DispatchArchTag<float>(params, stream);
  } else {
    UNIMPLEMENTED();
  }
}

class FusedMultiHeadAttentionInferenceKernel final : public user_op::OpKernel,
                                                     public user_op::CudaGraphSupport {
 public:
  FusedMultiHeadAttentionInferenceKernel() = default;
  ~FusedMultiHeadAttentionInferenceKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const Tensor* query = ctx->Tensor4ArgNameAndIndex("query", 0);
    const Tensor* key = ctx->Tensor4ArgNameAndIndex("key", 0);
    const Tensor* value = ctx->Tensor4ArgNameAndIndex("value", 0);
    const Tensor* attn_bias = nullptr;
    if (ctx->has_input("attn_bias", 0)) { attn_bias = ctx->Tensor4ArgNameAndIndex("attn_bias", 0); }
    Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    Tensor* tmp = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    const DataType data_type = query->data_type();
    CHECK_EQ(key->data_type(), data_type);
    CHECK_EQ(value->data_type(), data_type);
    CHECK_EQ(out->data_type(), data_type);
    const int64_t query_head_size = ctx->Attr<int64_t>("query_head_size");
    const bool causal = ctx->Attr<bool>("causal");
    const int64_t causal_diagonal_offset = ctx->Attr<int64_t>("causal_diagonal_offset");
    CHECK_GE(causal_diagonal_offset, 0);
    const std::string& query_layout = ctx->Attr<std::string>("query_layout");
    const std::string& key_layout = ctx->Attr<std::string>("key_layout");
    const std::string& value_layout = ctx->Attr<std::string>("value_layout");

    const auto ParseDims =
        [](const ShapeView& shape, const std::string& layout, const Optional<int64_t>& num_heads,
           const Optional<int64_t>& head_size, int64_t* b, int64_t* m, int64_t* h, int64_t* k,
           int64_t* b_stride, int64_t* m_stride, int64_t* h_stride) -> void {
      if (shape.NumAxes() == 3) {
        if (layout == "BM(HK)") {
          *b = shape.At(0);
          *m = shape.At(1);
          const int64_t hidden_size = shape.At(2);
          if (num_heads) {
            const int64_t expected_h = CHECK_JUST(num_heads);
            CHECK_EQ(hidden_size % expected_h, 0);
            *h = expected_h;
            *k = hidden_size / expected_h;
          } else if (head_size) {
            const int64_t expected_k = CHECK_JUST(head_size);
            CHECK_EQ(hidden_size % expected_k, 0);
            *h = hidden_size / expected_k;
            *k = expected_k;
          } else {
            UNIMPLEMENTED();
          }
          *h_stride = *k;
          *m_stride = *h_stride * *h;
          *b_stride = *m_stride * *m;
        } else if (layout == "MB(HK)") {
          *b = shape.At(1);
          *m = shape.At(0);
          const int64_t hidden_size = shape.At(2);
          if (num_heads) {
            const int64_t expected_h = CHECK_JUST(num_heads);
            CHECK_EQ(hidden_size % expected_h, 0);
            *h = expected_h;
            *k = hidden_size / expected_h;
          } else if (head_size) {
            const int64_t expected_k = CHECK_JUST(head_size);
            CHECK_EQ(hidden_size % expected_k, 0);
            *h = hidden_size / expected_k;
            *k = expected_k;
          } else {
            UNIMPLEMENTED();
          }
          *h_stride = *k;
          *b_stride = *h_stride * *h;
          *m_stride = *b_stride * *b;
        } else {
          UNIMPLEMENTED();
        }
      } else if (shape.NumAxes() == 4) {
        if (layout == "BMHK") {
          *b = shape.At(0);
          *m = shape.At(1);
          *h = shape.At(2);
          *k = shape.At(3);
          *h_stride = *k;
          *m_stride = *h_stride * *h;
          *b_stride = *m_stride * *m;
        } else if (layout == "BHMK") {
          *b = shape.At(0);
          *m = shape.At(2);
          *h = shape.At(1);
          *k = shape.At(3);
          *m_stride = *k;
          *h_stride = *m_stride * *m;
          *b_stride = *h_stride * *h;
        } else {
          UNIMPLEMENTED();
        }
        if (num_heads) {
          const int64_t expected_h = CHECK_JUST(num_heads);
          CHECK_EQ(*h, expected_h);
        }
        if (head_size) {
          const int64_t expected_k = CHECK_JUST(head_size);
          CHECK_EQ(*k, expected_k);
        }
      } else {
        UNIMPLEMENTED();
      };
    };

    int64_t q_b = 0;
    int64_t q_m = 0;
    int64_t q_h = 0;
    int64_t q_k = 0;
    int64_t q_b_stride = 0;
    int64_t q_m_stride = 0;
    int64_t q_h_stride = 0;
    ParseDims(query->shape_view(), query_layout, Optional<int64_t>(), query_head_size, &q_b, &q_m,
              &q_h, &q_k, &q_b_stride, &q_m_stride, &q_h_stride);

    int64_t k_b = 0;
    int64_t k_m = 0;
    int64_t k_h = 0;
    int64_t k_k = 0;
    int64_t k_b_stride = 0;
    int64_t k_m_stride = 0;
    int64_t k_h_stride = 0;
    ParseDims(key->shape_view(), key_layout, Optional<int64_t>(), query_head_size, &k_b, &k_m, &k_h,
              &k_k, &k_b_stride, &k_m_stride, &k_h_stride);
    CHECK_EQ(k_b, q_b);
    CHECK_EQ(k_h, q_h);

    int64_t v_b = 0;
    int64_t v_m = 0;
    int64_t v_h = 0;
    int64_t v_k = 0;
    int64_t v_b_stride = 0;
    int64_t v_m_stride = 0;
    int64_t v_h_stride = 0;
    ParseDims(value->shape_view(), value_layout, q_h, Optional<int64_t>(), &v_b, &v_m, &v_h, &v_k,
              &v_b_stride, &v_m_stride, &v_h_stride);
    CHECK_EQ(v_b, q_b);
    CHECK_EQ(v_m, k_m);
    CHECK_EQ(out->shape_view().NumAxes(), 3);
    CHECK_EQ(out->shape_view().At(0), q_b);
    CHECK_EQ(out->shape_view().At(1), q_m);
    CHECK_EQ(out->shape_view().At(2), q_h * v_k);

    auto* cuda_stream = ctx->stream()->As<ep::CudaStream>();

    // Compatible with typo `KERENL`
    const bool enable_trt_flash_attn =
        ParseBooleanFromEnv(
            "ONEFLOW_KERNEL_FMHA_ENABLE_TRT_FLASH_ATTN_IMPL",
            ParseBooleanFromEnv("ONEFLOW_KERENL_FMHA_ENABLE_TRT_FLASH_ATTN_IMPL", true))
        && ParseBooleanFromEnv("ONEFLOW_MATMUL_ALLOW_HALF_PRECISION_ACCUMULATION", false);
    const int arch = cuda_stream->cuda_arch() / 10;
    const bool is_trt_supported_arch = (arch == 75 || arch == 80 || arch == 86 || arch == 89);
    const bool is_trt_supported_head_size = ((q_k == 40) || (q_k == 64));
    // Avoid PackQKV overhead when seq_len is small.
    const bool is_long_seq_len = q_m >= 512;
    const bool is_trt_supported_layout = (query_layout == "BMHK" || query_layout == "BM(HK)")
                                         && (key_layout == "BMHK" || key_layout == "BM(HK)")
                                         && (value_layout == "BMHK" || value_layout == "BM(HK)");
    if (enable_trt_flash_attn && data_type == DataType::kFloat16 && q_m == k_m && q_k == v_k
        && is_trt_supported_head_size && is_long_seq_len && is_trt_supported_arch && (!causal)
        && attn_bias == nullptr && is_trt_supported_layout) {
      // The fmha implementation below is based on TensorRT's multiHeadFlashAttentionPlugin
      // implementation at:
      // https://github.com/NVIDIA/TensorRT/tree/main/plugin/multiHeadFlashAttentionPlugin
      int32_t cu_seqlens_d_size = (q_b + 1) * sizeof(int32_t);
      int32_t* cu_seqlens_d = reinterpret_cast<int32_t*>(tmp->mut_dptr());
      half* packed_qkv =
          reinterpret_cast<half*>(tmp->mut_dptr<char>() + GetCudaAlignedSize(cu_seqlens_d_size));
      constexpr int pack_size = 4;
      using PackType = Pack<half, pack_size>;
      const int64_t count = q_b * q_m * q_h * q_k * 3 / pack_size;
      PackQkv<PackType><<<(count - 1 + 256) / 256, 256, 0, cuda_stream->cuda_stream()>>>(
          q_b, q_m, q_h, q_k / pack_size, reinterpret_cast<const PackType*>(query->dptr()),
          reinterpret_cast<const PackType*>(key->dptr()),
          reinterpret_cast<const PackType*>(value->dptr()), reinterpret_cast<PackType*>(packed_qkv),
          cu_seqlens_d);

#ifdef WITH_CUDA_GRAPHS
      cudaStreamCaptureMode mode = cudaStreamCaptureModeRelaxed;
      if (cuda_stream->IsGraphCapturing()) {
        OF_CUDA_CHECK(cudaThreadExchangeStreamCaptureMode(&mode));
      }
#endif  // WITH_CUDA_GRAPHS
      nvinfer1::plugin::FusedMultiHeadFlashAttentionKernel const* kernels =
          nvinfer1::plugin::getFMHAFlashCubinKernels(nvinfer1::plugin::DATA_TYPE_FP16, arch);
#ifdef WITH_CUDA_GRAPHS
      if (cuda_stream->IsGraphCapturing()) {
        OF_CUDA_CHECK(cudaThreadExchangeStreamCaptureMode(&mode));
      }
#endif  // WITH_CUDA_GRAPHS
      nvinfer1::plugin::runFMHFAKernel(packed_qkv, cu_seqlens_d, out->mut_dptr(), q_b * q_m, arch,
                                       kernels, q_b, q_h, q_k, q_m, cuda_stream->cuda_stream());
      return;
    }

    Params params{};
    params.data_type = data_type;
    params.num_batches = q_b;
    params.num_heads = q_h;
    params.query_seq_len = q_m;
    params.kv_seq_len = k_m;
    params.head_size = q_k;
    params.value_head_size = v_k;
    params.q_stride_b = q_b_stride;
    params.q_stride_m = q_m_stride;
    params.q_stride_h = q_h_stride;
    params.k_stride_b = k_b_stride;
    params.k_stride_m = k_m_stride;
    params.k_stride_h = k_h_stride;
    params.v_stride_b = v_b_stride;
    params.v_stride_m = v_m_stride;
    params.v_stride_h = v_h_stride;
    params.query_ptr = query->dptr<char>();
    params.key_ptr = key->dptr<char>();
    params.value_ptr = value->dptr<char>();
    params.out_ptr = out->mut_dptr();
    const int64_t tmp_buffer_size = tmp->shape_view().elem_cnt();
    params.workspace = tmp->mut_dptr();
    params.workspace_size = tmp_buffer_size;
    params.causal = causal;
    params.causal_diagonal_offset = causal_diagonal_offset;
    if (attn_bias != nullptr) {
      const int64_t num_attn_bias_axes = attn_bias->shape_view().NumAxes();
      CHECK_GE(num_attn_bias_axes, 1);
      CHECK_LE(num_attn_bias_axes, 4);
      DimVector padded_attn_bias_shape;
      for (int i = 0; i < 4 - num_attn_bias_axes; ++i) { padded_attn_bias_shape.push_back(1); }
      for (int i = 0; i < num_attn_bias_axes; ++i) {
        padded_attn_bias_shape.push_back(attn_bias->shape_view().At(i));
      }
      CHECK_GE(padded_attn_bias_shape.at(3), k_m);
      int64_t bias_stride = padded_attn_bias_shape.at(3);
      if (padded_attn_bias_shape.at(2) == 1) {
        params.attn_bias_stride_m = 0;
      } else {
        CHECK_GE(padded_attn_bias_shape.at(2), q_m);
        params.attn_bias_stride_m = bias_stride;
        bias_stride *= padded_attn_bias_shape.at(2);
      }
      if (padded_attn_bias_shape.at(1) == 1) {
        params.attn_bias_stride_h = 0;
      } else {
        CHECK_EQ(padded_attn_bias_shape.at(1), q_h);
        params.attn_bias_stride_h = bias_stride;
        bias_stride *= q_h;
      }
      if (padded_attn_bias_shape.at(0) == 1) {
        params.attn_bias_stride_b = 0;
      } else {
        CHECK_EQ(padded_attn_bias_shape.at(0), q_b);
        params.attn_bias_stride_b = bias_stride;
      }
      params.attn_bias_ptr = attn_bias->dptr();
    } else {
      params.attn_bias_ptr = nullptr;
      params.attn_bias_stride_m = 0;
      params.attn_bias_stride_h = 0;
      params.attn_bias_stride_b = 0;
    }
    DispatchCutlassFmha(params, cuda_stream);
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

size_t InferTmpBufferSize(InferContext* ctx) {
  const auto& out_desc = ctx->OutputTensorDesc("out", 0);
  size_t buffer_size = 0;
  buffer_size +=
      GetCudaAlignedSize(out_desc.shape().elem_cnt() * GetSizeOfDataType(DataType::kFloat));
  buffer_size +=
      GetCudaAlignedSize(out_desc.shape().elem_cnt() * GetSizeOfDataType(out_desc.data_type())) * 3;
  buffer_size +=
      GetCudaAlignedSize((out_desc.shape().At(0) + 1) * GetSizeOfDataType(DataType::kInt32));
  return buffer_size;
}

}  // namespace

#define REGISTER_FUSED_MULTI_HEAD_ATTENTION_INFERENCE_KERNEL(dtype)    \
  REGISTER_USER_KERNEL("fused_multi_head_attention_inference")         \
      .SetCreateFn<FusedMultiHeadAttentionInferenceKernel>()           \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("out", 0) == dtype))   \
      .SetInferTmpSizeFn(InferTmpBufferSize);

REGISTER_FUSED_MULTI_HEAD_ATTENTION_INFERENCE_KERNEL(DataType::kFloat16)
REGISTER_FUSED_MULTI_HEAD_ATTENTION_INFERENCE_KERNEL(DataType::kFloat)

}  // namespace user_op

}  // namespace oneflow

#endif  // WITH_CUTLASS
